#include "MRTforce.cuh"
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

int main()
{
	const int Nstep = 1;
	int savepoint = 1;
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
		
	LBMpara params;
	params.Nx = 96;
	params.Ny = 104;
	params.Nz = 104;
	params.rho0 = 1.0;
	params.ux0 = 0.0;
	params.uy0 = 0.0;
	params.uz0 = 0.02;
	params.gravity = 0.00098;

	MRTGPU lbm;

	cout << "Simulation start!" << endl;
	lbm.init(params);

	cout << "Loop start!" << endl;
	hipEventRecord(start, 0);

	for (int step = 1; step <= Nstep; step++)
	{
		lbm.feq();
		lbm.rate_strain();
		lbm.mf_meq();
		lbm.Invf_feq();
		lbm.collision();
		lbm.boundary();
		lbm.macroscopic();
		if (step % savepoint == 0)
			lbm.output(step);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "Loop time is: " << time << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	lbm.freemem();

	hipDeviceReset();
	system("pause");
	return 0;
}