#include "hip/hip_runtime.h"
#include "MRTforce.cuh"
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <fstream>
#include <sstream>
#include <string>

using namespace std;

const int Q = 19;
__constant__ int cx[19];
__constant__ int cy[19];
__constant__ int cz[19];
__constant__ double w[19];
__constant__ int N[3];

//1.initialize
void MRTGPU::init(LBMpara params)
{
	Nx = params.Nx;
	Ny = params.Ny;
	Nz = params.Nz;
	rho0 = params.rho0;
	ux0 = params.ux0;
	uy0 = params.uy0;
	uz0 = params.uz0;
	gravity = params.gravity;
	int Nlattice = Nx * Ny * Nz;
	int QNlattice = Nx * Ny * Nz * Q;

	int _cx[19] = { 0,1,-1,0,0,0,0,1,-1,1,-1,1,-1,1,-1,0,0,0,0 };
	int _cy[19] = { 0,0,0,1,-1,0,0,1,1,-1,-1,0,0,0,0,1,-1,1,-1 };
	int _cz[19] = { 0,0,0,0,0,1,-1,0,0,0,0,1,1,-1,-1,1,1,-1,-1 };//Yu's setting
	double _w[19] = { 1.0 / 3.0,1.0 / 18.0,1.0 / 18.0,1.0 / 18.0,1.0 / 18.0,1.0 / 18.0,1.0 / 18.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,
		1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0 };
	int _N[3] = { Nx,Ny,Nz };

	hipMemcpyToSymbol(HIP_SYMBOL(cx), _cx, sizeof(int) * Q);
	hipMemcpyToSymbol(HIP_SYMBOL(cy), _cy, sizeof(int) * Q);
	hipMemcpyToSymbol(HIP_SYMBOL(cz), _cz, sizeof(int) * Q);
	hipMemcpyToSymbol(HIP_SYMBOL(w), _w, sizeof(double) * Q);
	hipMemcpyToSymbol(HIP_SYMBOL(N), _N, sizeof(int) * 3);

	int threadsAlongX = 8;
	int threadsAlongY = 8;
	int threadsAlongZ = 8;

	block = dim3(threadsAlongX, threadsAlongY, threadsAlongZ);
	grid = dim3(1 + (Nx - 1) / threadsAlongX, 1 + (Ny - 1) / threadsAlongY, 1 + (Nz - 1) / threadsAlongZ);

	// allocate memory on CPU and GPU 
	h_geo = (int*)malloc(sizeof(int) * Nlattice);
	h_ux = (double*)malloc(sizeof(double) * Nlattice);
	h_uy = (double*)malloc(sizeof(double) * Nlattice);
	h_uz = (double*)malloc(sizeof(double) * Nlattice);
	h_rho = (double*)malloc(sizeof(double) * Nlattice);	
	h_f = (double*)malloc(sizeof(double) * Nlattice * Q);
	h_f_post = (double*)malloc(sizeof(double) * Nlattice * Q);

	output_rho = (double*)malloc(sizeof(double) * Nlattice);
	output_ux = (double*)malloc(sizeof(double) * Nlattice);
	output_uy = (double*)malloc(sizeof(double) * Nlattice);
	output_uz = (double*)malloc(sizeof(double) * Nlattice);

	hipMalloc((void**)&d_geo, Nlattice * sizeof(int));
	hipMalloc((void**)&d_f, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&d_f_post, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&d_feq, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&m_f, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&m_eq, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&inv_f, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&inv_feq, sizeof(double) * Nlattice * Q);

	hipMalloc((void**)&d_rho, Nlattice * sizeof(double));
	hipMalloc((void**)&d_ux, Nlattice * sizeof(double));
	hipMalloc((void**)&d_uy, Nlattice * sizeof(double));
	hipMalloc((void**)&d_uz, Nlattice * sizeof(double));
	
	hipMalloc((void**)&s_xx, Nlattice * sizeof(double));
	hipMalloc((void**)&s_xy, Nlattice * sizeof(double));
	hipMalloc((void**)&s_xz, Nlattice * sizeof(double));
	hipMalloc((void**)&s_yx, Nlattice * sizeof(double));
	hipMalloc((void**)&s_yy, Nlattice * sizeof(double));
	hipMalloc((void**)&s_yz, Nlattice * sizeof(double));
	hipMalloc((void**)&s_zx, Nlattice * sizeof(double));
	hipMalloc((void**)&s_zy, Nlattice * sizeof(double));
	hipMalloc((void**)&s_zz, Nlattice * sizeof(double));
	hipMalloc((void**)&sigma, Nlattice * sizeof(double));

	ifstream inf("D:\\Structrue-96.dat"); //read the geo file

	int i = 0;

	while (inf >> h_geo[i])
		++i;
	inf.close();

	for (int z = 0; z < Nz; z++)
	{
		for (int y = 0; y < Ny; y++)
		{
			for (int x = 0; x < Nx; x++)
			{
				int index = z * Nx * Ny + y * Nx + x;

				if (h_geo[index] == 0 || h_geo[index] == 3 || h_geo[index] == 4)
				{
					h_ux[index] = ux0;
					h_uy[index] = uy0;
					h_uz[index] = uz0 - gravity / 2;					
					h_rho[index] = rho0;
				}
				else if (h_geo[index] == 1)
				{
					h_ux[index] = 0.0;
					h_uy[index] = 0.0;
					h_uz[index] = 0.0;					
					h_rho[index] = 0.0;
				}
				else if (h_geo[index] == 2 || h_geo[index] == 5 || h_geo[index] == 6)
				{
					h_ux[index] = 0.0;
					h_uy[index] = 0.0;
					h_uz[index] = 0.0;					
					h_rho[index] = rho0;
				}
			}
		}
	}
	for (int z = 0; z < Nz; z++)
	{
		for (int y = 0; y < Ny; y++)
		{
			for (int x = 0; x < Nx; x++)
			{
				int index = z * Nx * Ny + y * Nx + x;

				for (int q = 0; q < Q; q++)
				{
					if (h_geo[index] == 0 || h_geo[index] == 2 || h_geo[index] == 3 || h_geo[index] == 4 || h_geo[index] == 5 || h_geo[index] == 6)
					{
						h_f[Nlattice * q + index] = _w[q] * h_rho[index] * (1 + 3.0 * (_cx[q] * h_ux[index] + _cy[q] * h_uy[index] + _cz[q] * h_uz[index])
							+ 4.5 * (_cx[q] * h_ux[index] + _cy[q] * h_uy[index] + _cz[q] * h_uz[index]) * (_cx[q] * h_ux[index] + _cy[q] * h_uy[index] + _cz[q] * h_uz[index])
							- 1.5 * (h_ux[index] * h_ux[index] + h_uy[index] * h_uy[index] + h_uz[index] * h_uz[index]));
						h_f_post[Nlattice * q + index] = 0.0;
					}
					else if (h_geo[index] == 1)
					{
						h_f[Nlattice * q + index] = 0.0;
						h_f_post[Nlattice * q + index] = 0.0;
					}
				}
			}
		}
	}
	hipMemcpy(d_geo, h_geo, Nlattice * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_f, h_f, QNlattice * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_f_post, h_f_post, QNlattice * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_ux, h_ux, Nlattice * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_uy, h_uy, Nlattice * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_uz, h_uz, Nlattice * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_rho, h_rho, Nlattice * sizeof(double), hipMemcpyHostToDevice);		
}

//2.gpu side functions
__global__ void kernelfeq(int* __restrict__ d_geo, double* __restrict__ d_feq,
	double* __restrict__ d_rho, double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz);

__global__ void kernelrate_strain(int* __restrict__ d_geo, double* __restrict__ d_feq, double* __restrict__ d_f,
	double* __restrict__ s_xx, double* __restrict__ s_xy, double* __restrict__ s_xz, 
	double* __restrict__ s_yx, double* __restrict__ s_yy, double* __restrict__ s_yz,
	double* __restrict__ s_zx, double* __restrict__ s_zy, double* __restrict__ s_zz, double* __restrict__ sigma);

__global__ void kernelmf_meq(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ m_f, double* __restrict__ m_eq,
	double* __restrict__ sigma, double* __restrict__ d_rho, double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz);

__global__ void kernelInvf_feq(int* __restrict__ d_geo, double* __restrict__ m_f, double* __restrict__ m_eq, 
	double* __restrict__ inv_f, double* __restrict__ inv_feq);

__global__ void kernelcollision(int* __restrict__ d_geo, double* __restrict__ inv_f, double* __restrict__ inv_feq, 
	double* __restrict__ d_f, double* __restrict__ d_f_post);

__global__ void kernelboundary(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ d_rho);

__global__ void kernelmacroscopic(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ d_rho,
	double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz);

//3.class functions
void MRTGPU::feq()
{
	kernelfeq << <grid, block >> > (d_geo, d_feq, d_rho, d_ux, d_uy, d_uz);
	hipDeviceSynchronize();
}
void MRTGPU::rate_strain()
{
	kernelrate_strain << <grid, block >> > (d_geo, d_feq, d_f, s_xx, s_xy, s_xz, s_yx, s_yy, s_yz, s_zx, s_zy, s_zz, sigma);
	hipDeviceSynchronize();
}
void MRTGPU::mf_meq()
{
	kernelmf_meq << <grid, block >> > (d_geo, d_f, m_f, m_eq, sigma, d_rho, d_ux, d_uy, d_uz);
	hipDeviceSynchronize();
}
void MRTGPU::Invf_feq()
{
	kernelInvf_feq << <grid, block >> > (d_geo, m_f, m_eq, inv_f, inv_feq);
	hipDeviceSynchronize();
}
void MRTGPU::collision()
{
	kernelcollision << <grid, block >> > (d_geo, inv_f, inv_feq, d_f, d_f_post);
	hipDeviceSynchronize();
}
void MRTGPU::boundary()
{
	kernelboundary << <grid, block >> > (d_geo, d_f, d_rho);
	hipDeviceSynchronize();
}
void MRTGPU::macroscopic()
{
	kernelmacroscopic << <grid, block >> > (d_geo, d_f, d_rho, d_ux, d_uy, d_uz);
	hipDeviceSynchronize();
}

//4.gpu functions definition
__global__ void kernelfeq(int* __restrict__ d_geo, double* __restrict__ d_feq,
	double* __restrict__ d_rho, double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz)
{
	const double rho0 = 1.0;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	for (int q = 0; q < 19; q++)
	{
		if (d_geo[index] == 0 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4 || d_geo[index] == 5 || d_geo[index] == 6)
			d_feq[index + Nlattice * q] = w[q] * (d_rho[index] + rho0 * (3.0 * (cx[q] * d_ux[index] + cy[q] * d_uy[index] + cz[q] * d_uz[index])
				+ 4.5 * (cx[q] * d_ux[index] + cy[q] * d_uy[index] + cz[q] * d_uz[index]) * (cx[q] * d_ux[index] + cy[q] * d_uy[index] + cz[q] * d_uz[index])
				- 1.5 * (d_ux[index] * d_ux[index] + d_uy[index] * d_uy[index] + d_uz[index] * d_uz[index])));
		else if (d_geo[index] == 1)
			d_feq[index + Nlattice * q] = 0.0;
	}
}

__global__ void kernelrate_strain(int* __restrict__ d_geo, double* __restrict__ d_feq, double* __restrict__ d_f,
	double* __restrict__ s_xx, double* __restrict__ s_xy, double* __restrict__ s_xz,
	double* __restrict__ s_yx, double* __restrict__ s_yy, double* __restrict__ s_yz,
	double* __restrict__ s_zx, double* __restrict__ s_zy, double* __restrict__ s_zz, double* __restrict__ sigma)
{
	const int Q = 19;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	s_xx[index] = 0.0; s_xy[index] = 0.0; s_xz[index] = 0.0;
	s_yx[index] = 0.0; s_yy[index] = 0.0; s_yz[index] = 0.0;
	s_zx[index] = 0.0; s_zy[index] = 0.0; s_zz[index] = 0.0;

	for (int q = 0; q < Q; q++)
	{
		s_xx[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cx[q] * cx[q];
		s_xy[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cx[q] * cy[q];
		s_xz[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cx[q] * cz[q];
		s_yx[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cx[q] * cy[q];
		s_yy[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cy[q] * cy[q];
		s_yz[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cy[q] * cz[q];
		s_zx[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cx[q] * cz[q];
		s_zy[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cy[q] * cz[q];
		s_zz[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cz[q] * cz[q];
	}
	if (d_geo[index] == 0 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4 || d_geo[index] == 5 || d_geo[index] == 6)
		sigma[index] = sqrt(2 * (s_xx[index] * s_xx[index] + s_xy[index] * s_xy[index] + s_xz[index] * s_xz[index]
			+ s_yx[index] * s_yx[index] + s_yy[index] * s_yy[index] + s_yz[index] * s_yz[index]
			+ s_zx[index] * s_zx[index] + s_zy[index] * s_zy[index] + s_zz[index] * s_zz[index]));
	else if (d_geo[index] == 1)
		sigma[index] = 0.0;
}

__global__ void kernelmf_meq(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ m_f, double* __restrict__ m_eq, 
	double* __restrict__ sigma,	double* __restrict__ d_rho, double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz)
{
	const double rho0 = 1.0;
	const double tau = 0.55;/////////////calculaton
	const double C_Smag = 0.16;

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	const double S_diag[19] = { 0.0, 1.19,1.4,0.0,1.2,0.0,1.2,0.0,1.2,1.0,1.4,1.0,1.4,1.0,1.0,1.0,1.98,1.98,1.98 };
	const double F_diag[19] = { 19.0,2394.0,252.0,10.0,40.0,10.0,40.0,10.0,40.0,36.0,72.0,12.0,24.0,4.0,4.0,4.0,8.0,8.0,8.0 };

	const double mm[19][19] = {
	{1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0},
	{-30.0,-11.0,-11.0,-11.0,-11.0,-11.0,-11.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0},
	{12.0,-4.0,-4.0,-4.0,-4.0,-4.0,-4.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0},
	{0.0,1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,-4.0,4.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,1.0,-1.0,0.0,0.0,1.0,1.0,-1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,-4.0,4.0,0.0,0.0,1.0,1.0,-1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,1.0,-1.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,-1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,-4.0,4.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,-1.0,-1.0},
	{0.0,2.0,2.0,-1.0,-1.0,-1.0,-1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,-2.0,-2.0,-2.0,-2.0},
	{0.0,-4.0,-4.0,2.0,2.0,2.0,2.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,-2.0,-2.0,-2.0,-2.0},
	{0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,-2.0,-2.0,2.0,2.0,1.0,1.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,-1.0,1.0,-1.0,1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,-1.0,-1.0,1.0,1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,1.0,1.0} };

	for (int q = 0; q < 19; q++)
	{
		m_f[index + Nlattice * q] = 0.0;
		m_eq[index + Nlattice * q] = 0.0;
	}
	if (d_geo[index] == 0 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4 || d_geo[index] == 5 || d_geo[index] == 6)
	{
		for (int q = 0; q < Q; q++)
		{
			for (int qq = 0; qq < Q; qq++)
			{
				if (q == 0 || q == 1 || q == 2 || q == 3 || q == 4 || q == 5 || q == 6 || q == 7 || q == 8 || q == 10 || q == 12 || q == 16 || q == 17 || q == 18)
					m_f[index + Nlattice * q] += S_diag[q] / F_diag[q] * mm[q][qq] * d_f[index + Nlattice * qq];
				else if (q == 9 || q == 11 || q == 13 || q == 14 || q == 15)
					m_f[index + Nlattice * q] += mm[q][qq] * d_f[index + Nlattice * qq] / (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[q];
			}//tau_t = 0.5*(pow(pow(tau,2) + 18.0*pow(C_Smagorinsky,2)*sigma[index],0.5) - tau);
		}
		m_eq[index + Nlattice * 0] = S_diag[0] / F_diag[0] * d_rho[index];
		m_eq[index + Nlattice * 1] = S_diag[1] / F_diag[1] * ((-11.0) * d_rho[index] + 19.0 * (d_ux[index] * d_ux[index] + d_uy[index] * d_uy[index] + d_uz[index] * d_uz[index]));
		m_eq[index + Nlattice * 2] = S_diag[2] / F_diag[2] * (-475.0 / 63.0) * (d_ux[index] * d_ux[index] + d_uy[index] * d_uy[index] + d_uz[index] * d_uz[index]);
		m_eq[index + Nlattice * 3] = S_diag[3] / F_diag[3] * d_ux[index];
		m_eq[index + Nlattice * 4] = S_diag[4] / F_diag[4] * (-2.0 / 3.0) * d_ux[index];
		m_eq[index + Nlattice * 5] = S_diag[5] / F_diag[5] * d_uy[index];
		m_eq[index + Nlattice * 6] = S_diag[6] / F_diag[6] * (-2.0 / 3.0) * d_uy[index];
		m_eq[index + Nlattice * 7] = S_diag[7] / F_diag[7] * d_uz[index];
		m_eq[index + Nlattice * 8] = S_diag[8] / F_diag[8] * (-2.0 / 3.0) * d_uz[index];
		m_eq[index + Nlattice * 9] = 2.0 * d_ux[index] * d_ux[index] - (d_uy[index] * d_uy[index] + d_uz[index] * d_uz[index])
			/ (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[9];
		m_eq[index + Nlattice * 10] = 0.0;
		m_eq[index + Nlattice * 11] = (d_uy[index] * d_uy[index] - d_uz[index] * d_uz[index])
			/ (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[11];
		m_eq[index + Nlattice * 12] = 0.0;
		m_eq[index + Nlattice * 13] = d_ux[index] * d_uy[index]
			/ (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[13];
		m_eq[index + Nlattice * 14] = d_uy[index] * d_uz[index]
			/ (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[14];
		m_eq[index + Nlattice * 15] = d_ux[index] * d_uz[index]
			/ (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[15];
		m_eq[index + Nlattice * 16] = 0.0;
		m_eq[index + Nlattice * 17] = 0.0;
		m_eq[index + Nlattice * 18] = 0.0;
	}
	if (d_geo[index] == 1)
	{
		for (int q = 0; q < 19; q++)
		{
			m_f[index + Nlattice * q] = 0.0;
			m_eq[index + Nlattice * q] = 0.0;
		}
	}
}

__global__ void kernelInvf_feq(int* __restrict__ d_geo, double* __restrict__ m_f, double* __restrict__ m_eq,
	double* __restrict__ inv_f, double* __restrict__ inv_feq)
{
	const int Q = 19;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	const double mmt[19][19] = {
	{1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0},
	{-30.0,-11.0,-11.0,-11.0,-11.0,-11.0,-11.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0},
	{12.0,-4.0,-4.0,-4.0,-4.0,-4.0,-4.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0},
	{0.0,1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,-4.0,4.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,1.0,-1.0,0.0,0.0,1.0,1.0,-1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,-4.0,4.0,0.0,0.0,1.0,1.0,-1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,1.0,-1.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,-1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,-4.0,4.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,-1.0,-1.0},
	{0.0,2.0,2.0,-1.0,-1.0,-1.0,-1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,-2.0,-2.0,-2.0,-2.0},
	{0.0,-4.0,-4.0,2.0,2.0,2.0,2.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,-2.0,-2.0,-2.0,-2.0},
	{0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,-2.0,-2.0,2.0,2.0,1.0,1.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,-1.0,1.0,-1.0,1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,-1.0,-1.0,1.0,1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,1.0,1.0} };

	for (int q = 0; q < Q; q++)
	{
		inv_f[index + Nlattice * q] = 0.0;
		inv_feq[index + Nlattice * q] = 0.0;
	}
	if (d_geo[index] == 0 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4 || d_geo[index] == 5 || d_geo[index] == 6)
	{
		for (int q = 0; q < Q; q++)
		{
			for (int qq = 0; qq < Q; qq++)
			{
				inv_f[index + Nlattice * q] += mmt[qq][q] * m_f[index + Nlattice * qq];
				inv_feq[index + Nlattice * q] += mmt[qq][q] * m_eq[index + Nlattice * qq];
			}
		}
	}
}

__global__ void kernelcollision(int* __restrict__ d_geo, double* __restrict__ inv_f, double* __restrict__ inv_feq,
	double* __restrict__ d_f, double* __restrict__ d_f_post)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];
	int indexf[19];
	const double gravity = 0.00098;

	for (int q = 0; q < 19; q++)
	{
		int i_1 = (i + cx[q] + N[0]) % N[0];
		int j_1 = (j + cy[q] + N[1]) % N[1];
		int k_1 = (k + cz[q] + N[2]) % N[2];
		indexf[q] = k_1 * N[0] * N[1] + j_1 * N[0] + i_1;

		if (d_geo[index] == 0 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4 || d_geo[index] == 5 || d_geo[index] == 6)
			d_f_post[indexf[q] + Nlattice * q] = d_f[index + Nlattice * q] - (inv_f[index + Nlattice * q] - inv_feq[index + Nlattice * q])
			- 3.0 * w[q] * (cz[q] * gravity);

		else if (d_geo[index] == 1)
			d_f_post[indexf[q] + Nlattice * q] = 0.0;
	}

	for (int q = 0; q < 19; q++) //swap
	{
		d_f[index + Nlattice * q] = d_f_post[index + Nlattice * q];
		d_f[index + Nlattice * q] = d_f_post[index + Nlattice * q];
	}
}

__global__ void kernelboundary(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ d_rho)
{
	const double u0 = 0.02 - 0.00098 / 2;///////////////////////////////////calculation
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int index_out = (k - 1) * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];
	int indexf[19];

	for (int q = 0; q < 19; q++)
	{
		int i_1 = (i + cx[q] + N[0]) % N[0];
		int j_1 = (j + cy[q] + N[1]) % N[1];
		int k_1 = (k + cz[q] + N[2]) % N[2];
		indexf[q] = k_1 * N[0] * N[1] + j_1 * N[0] + i_1;

		if (d_geo[index] == 2 || d_geo[index] == 5 || d_geo[index] == 6)
		{
			if (d_geo[indexf[q]] == 1)
			{
				if (q == 1) d_f[index + Nlattice * 2] = d_f[index + Nlattice * q];
				else if (q == 2) d_f[index + Nlattice * 1] = d_f[index + Nlattice * q];
				else if (q == 3) d_f[index + Nlattice * 4] = d_f[index + Nlattice * q];
				else if (q == 4) d_f[index + Nlattice * 3] = d_f[index + Nlattice * q];
				else if (q == 5) d_f[index + Nlattice * 6] = d_f[index + Nlattice * q];
				else if (q == 6) d_f[index + Nlattice * 5] = d_f[index + Nlattice * q];
				else if (q == 7) d_f[index + Nlattice * 10] = d_f[index + Nlattice * q];
				else if (q == 8) d_f[index + Nlattice * 9] = d_f[index + Nlattice * q];
				else if (q == 9) d_f[index + Nlattice * 8] = d_f[index + Nlattice * q];
				else if (q == 10) d_f[index + Nlattice * 7] = d_f[index + Nlattice * q];
				else if (q == 11) d_f[index + Nlattice * 14] = d_f[index + Nlattice * q];
				else if (q == 12) d_f[index + Nlattice * 13] = d_f[index + Nlattice * q];
				else if (q == 13) d_f[index + Nlattice * 12] = d_f[index + Nlattice * q];
				else if (q == 14) d_f[index + Nlattice * 11] = d_f[index + Nlattice * q];
				else if (q == 15) d_f[index + Nlattice * 18] = d_f[index + Nlattice * q];
				else if (q == 16) d_f[index + Nlattice * 17] = d_f[index + Nlattice * q];
				else if (q == 17) d_f[index + Nlattice * 16] = d_f[index + Nlattice * q];
				else if (q == 18) d_f[index + Nlattice * 15] = d_f[index + Nlattice * q];
			}
		}
		else if (d_geo[index] == 3)
		{
			if (q == 6)
				d_f[index + Nlattice * 5] = d_f[index + Nlattice * q] + u0 * d_rho[index] / 3.0;
			else if (q == 13)
				d_f[index + Nlattice * 12] = d_f[index + Nlattice * q] + u0 * d_rho[index] / 6.0
				- 0.5 * (cx[12] * (d_f[index + Nlattice * 1] - d_f[index + Nlattice * 2])
					+ cy[12] * (d_f[index + Nlattice * 3] - d_f[index + Nlattice * 4]));
			else if (q == 14)
				d_f[index + Nlattice * 11] = d_f[index + Nlattice * q] + u0 * d_rho[index] / 6.0
				- 0.5 * (cx[11] * (d_f[index + Nlattice * 1] - d_f[index + Nlattice * 2])
					+ cy[11] * (d_f[index + Nlattice * 3] - d_f[index + Nlattice * 4]));
			else if (q == 17)
				d_f[index + Nlattice * 16] = d_f[index + Nlattice * q] + u0 * d_rho[index] / 6.0
				- 0.5 * (cx[16] * (d_f[index + Nlattice * 1] - d_f[index + Nlattice * 2])
					+ cy[16] * (d_f[index + Nlattice * 3] - d_f[index + Nlattice * 4]));
			else if (q == 18)
				d_f[index + Nlattice * 15] = d_f[index + Nlattice * q] + u0 * d_rho[index] / 6.0
				- 0.5 * (cx[15] * (d_f[index + Nlattice * 1] - d_f[index + Nlattice * 2])
					+ cy[15] * (d_f[index + Nlattice * 3] - d_f[index + Nlattice * 4]));
		}
		else if (d_geo[index] == 4)
		{
			if (q == 5)
				d_f[index + Nlattice * 6] = d_f[index_out + Nlattice * 6];
			else if (q == 11)
				d_f[index + Nlattice * 14] = d_f[index_out + Nlattice * 14];
			else if (q == 12)
				d_f[index + Nlattice * 13] = d_f[index_out + Nlattice * 13];
			else if (q == 15)
				d_f[index + Nlattice * 18] = d_f[index_out + Nlattice * 18];
			else if (q == 16)
				d_f[index + Nlattice * 17] = d_f[index_out + Nlattice * 17];
		}
	}
}

__global__ void kernelmacroscopic(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ d_rho,
	double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz)
{
	const double rho0 = 1.0;
	const double gravity = 0.00098;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];
	
	d_rho[index] = 0.0;
	d_ux[index] = 0.0;
	d_uy[index] = 0.0;
	d_uz[index] = 0.0;

	if (d_geo[index] == 0 || d_geo[index] == 3|| d_geo[index] == 4)
	{
		for (int q = 0; q < 19; q++)
		{
			d_rho[index] += d_f[index + Nlattice * q];
			d_ux[index] += cx[q] * d_f[index + Nlattice * q];
			d_uy[index] += cy[q] * d_f[index + Nlattice * q];
			d_uz[index] += cz[q] * d_f[index + Nlattice * q];
		}
		d_uz[index] += d_uz[index] - gravity / 2;
	}
	else if (d_geo[index] == 2 || d_geo[index] == 5 || d_geo[index] == 6)
	{
		for (int q = 0; q < 19; q++)
		{
			d_rho[index] += d_f[index + Nlattice * q];
			d_ux[index] = 0.0;
			d_uy[index] = 0.0;
			d_uz[index] = 0.0;//no slip wall
		}
	}
	else if (d_geo[index] == 1)
	{
		d_rho[index] = 0.0;
		d_ux[index] = 0.0;
		d_uy[index] = 0.0;
		d_uz[index] = 0.0;
	}	
}

void MRTGPU::output(int t)
{
	ofstream outputfile;
	stringstream sfile;
	sfile << "D:\\MRTforce-" << t << ".dat";
	string datafilename = sfile.str();

	outputfile.open(datafilename.c_str());
	outputfile << "TITLE = \"Case Data\"" << endl;
	outputfile << "VARIABLES = \"X\", \"Y\", \"Z\", \"Density\", \"Uz\"" << endl;
	outputfile << "ZONE I = 96, J = 104, K = 104, DATAPACKING = POINT" << endl;

	int Nlattice = Nx * Ny * Nz;
	hipMemcpy(output_rho, d_rho, Nlattice * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(output_uz, d_uz, Nlattice * sizeof(double), hipMemcpyDeviceToHost);

	for (int z = 0; z < Nz; z++)
	{
		for (int y = 0; y < Ny; y++)
		{
			for (int x = 0; x < Nx; x++)
			{
				int k = z * Nx * Ny + y * Nx + x;
				outputfile << setprecision(4) << x + 1 << "\t" << y + 1 << "\t " << z + 1 << "\t " << output_rho[k] << "\t " << output_uz[k] << endl;
			}
		}
	}
	outputfile.close();
}

void MRTGPU::freemem()
{
	free(h_geo);
	free(h_f);
	free(h_f_post);
	free(h_rho);
	free(h_ux);
	free(h_uy);
	free(h_uz);
	free(output_rho);
	free(output_ux);
	free(output_uy);
	free(output_uz);

	hipFree(d_geo);
	hipFree(d_f);
	hipFree(d_f_post);	hipFree(d_feq);
	hipFree(m_f); hipFree(m_eq);
	hipFree(inv_f); hipFree(inv_feq);
	hipFree(d_rho);
	hipFree(d_ux);
	hipFree(d_uy);
	hipFree(d_uz);
	hipFree(s_xx); hipFree(s_xy); hipFree(s_xz);
	hipFree(s_yx); hipFree(s_yy); hipFree(s_yz);
	hipFree(s_zx); hipFree(s_zy); hipFree(s_zz);
	hipFree(sigma);
}